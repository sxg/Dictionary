
#include <hip/hip_runtime.h>
int getRowMajorIndex(const int colMajorIndex, const int width, const int height) {
	int row = colMajorIndex % height;
	int col = colMajorIndex / height;
	return row * width + col;
}

int getColMajorIndex(const int rowMajorIndex, const int width, const int height) {
	return getRowMajorIndex(rowMajorIndex, height, width);
}